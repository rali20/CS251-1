
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#include<pthread.h>
#include<math.h>

#define MAX_THREAD 1024

#define USAGE_EXIT(s) do{ \
                            printf("Usage: %s <# of elements> <random seed> \n %s\n", argv[0], s); \
                            exit(-1);\
                    }while(0);

#define CUDA_ERROR_EXIT(str) do{\
                                hipError_t err = hipGetLastError();\
                                if( err != hipSuccess){\
                                         printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                         exit(-1);\
                                }\
                             }while(0);

#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

__global__ void calculate(long long int *da, const long long int max_in, const long long int in, const long long int prev, long long int *d_ans, const long long int ctr, const long long int u){
      long long int i = blockDim.x * blockIdx.x + threadIdx.x;
      long long int l =  i*in + prev;
      long long int r =  (i+1)*in-1 + prev;
      if(r > max_in)
           return;
      if(i%2==0){
        da[l]=da[l]^da[r];
      }
      else{
        da[r]=da[l]^da[r];
      }
      if(prev!=0 && ctr==u) da[0] = da[0]^da[prev];
      if(ctr==u)
        *d_ans = da[0];
}

int main(int argc, char **argv)
{
  struct timeval start, end;
  long long int *a, num_elements, ctr;
  long long int *ans;
  long long int *d_ans;

  if(argc !=3)
           USAGE_EXIT("not enough parameters");

  num_elements = atoi(argv[1]);
  if(num_elements <=0)
          USAGE_EXIT("invalid num elements");

  long long int SEED = atoi(argv[2]);
  long long int size = num_elements * sizeof(long long int);

  a = (long long int *)malloc(size);
  ans = (long long int *)malloc(sizeof(long long int));
  if(!a){
          USAGE_EXIT("invalid num elements, not enough memory");
  }

  srand(SEED);

  for(ctr=0; ctr<num_elements; ++ctr)
        a[ctr] = random();
  long long int * da;
  hipMalloc(&da,  size);
  CUDA_ERROR_EXIT("cudaMalloc1");

  hipMalloc(&d_ans, sizeof(long long int));
  CUDA_ERROR_EXIT("cudaMalloc2");

  hipMemcpy(da, a, size, hipMemcpyHostToDevice);
  CUDA_ERROR_EXIT("memcpy1");

  gettimeofday(&start, NULL);
  long long int prev = 0;
  long long int max_in = 0;
  long long int flag=0;
  long long int num = num_elements;
  for(;;){
    if(num_elements==1){
      flag=1;
      break;
    }
    if(num_elements<=0)
      break;
    long long int u = (long long int)log2((double)num_elements);
    long long int x = pow(2,u);
    max_in = prev + x -1;
    for(ctr=1; ctr<=u; ++ctr){
      long long int in = pow(2,ctr);
      long long int threads = num_elements/in;
      if(threads>MAX_THREAD)threads = MAX_THREAD;
      long long int blocks = num_elements/threads;
      calculate<<<blocks, threads>>>(da, max_in, in, prev, d_ans, ctr, u);
      CUDA_ERROR_EXIT("kernel invocation");
    }
    prev += pow(2,u);
    num_elements = num - prev;
  }

  hipMemcpy(ans, d_ans, sizeof(long long int), hipMemcpyDeviceToHost);
  CUDA_ERROR_EXIT("memcpy2");

  if(flag==1 && num==1){
    (*ans) = 0^a[num-1];
  }
  else if(flag==1){
    (*ans) = (*ans)^a[num-1];
  }
  printf("XOR = %lld\n", (*ans));

  gettimeofday(&end, NULL);
  printf("Time taken = %ld microsecs\n", TDIFF(start, end));
  free(a);
  hipFree(da);
  free(ans);
  hipFree(d_ans);
}
